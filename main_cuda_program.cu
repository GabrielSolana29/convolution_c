#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


// a contains the image, b contains the kernel filter, and r is the matrix where the answer will be returned
__global__ void kernelConvolution(double *a, double *b, double *r, int size_col, int size_row, int size, int kernel_size) {
    int i = threadIdx.x+blockDim.x*blockIdx.x;
    int j = threadIdx.y+blockDim.y*blockIdx.y;
    int k,l;        
    int pos = i+(j*size_col);
    double sum = 0;        
    int cont = 0;
    
    // Iterate throught the kernel filter
    for (l = 0; l < kernel_size; l++){              
        for (k = 0; k < kernel_size; k++){                              
            sum += a[(pos+l)+(k*size_row)] * b[cont];          
            cont++;
        }
    }    
    // Save the value of the pixel in the vector r "result"
    r[i+(j*size_row)] = sum;         
}

float convolutionDeviceKernel(double *a, double *b, double *r, int size_col, int size_row, int kernel_size) {
    double *aD, *bD,*rD;    
    int size=size_col*size_row*sizeof(double); 
    int size_k = kernel_size*kernel_size*sizeof(double);
    hipEvent_t start, stop; 
    float t; 
    
    // Define the dimensions of the blocks and kernels
    dim3 bloques((int)floor((double)size_row/kernel_size*1.0),(int)floor((double)size_col/kernel_size*1.0));     
    dim3 hilos(kernel_size,kernel_size); 
    
    hipEventCreate(&start); hipEventCreate(&stop); 
    
    hipMalloc(&aD, size);hipMalloc(&rD, size); hipMalloc(&bD, size_k); 
    
    // Obtain the values for a and b from the host
    hipMemcpy(aD, a, size, hipMemcpyDefault);    
    hipMemcpy(bD, b, size_k , hipMemcpyDefault);     
    
    // Start counting the execution time
    hipEventRecord(start, 0); 
    
    // Convolucion
    kernelConvolution<<<bloques, hilos>>>(aD, bD, rD, size_col,size_row,size,kernel_size);
    
    // Stop counting the execution time       
    hipEventRecord(stop, 0); 
    
    // Copy to host 
    hipMemcpy(a, aD,size, hipMemcpyDefault);
    hipMemcpy(b, bD,size_k, hipMemcpyDefault);
    hipMemcpy(r, rD,size, hipMemcpyDefault);
  
    // Free memory
    hipFree(aD); hipFree(bD); hipFree(rD); 
     
    hipEventSynchronize(stop);

    hipEventElapsedTime(&t, start, stop); 
    hipEventDestroy(start); hipEventDestroy(stop); 
    // Return the time elapsed
    return t; 
}

////////////// MAIN ///////////////////////////////////

int main(int argc, char **argv) {
    int size_col;
    int size_row; 
    int d=0, kl=0;     
    int size_k = 9*sizeof(double);
    float t; 
  	int i,j;
  	double *tmp;
  	double *tmp2;
    // a=image, b=kernel filter, r = result image
    double *a,*b,*r;
   	// Select the parameters
	  int image_select = 2;
	  int kernel_select = 3;    
    int noIterations = 10;
      
   // Receive parameters from the console   
  	if (argc > 1){
     d = atoi(argv[1]);
     } 
    if (argc > 2){ 
     image_select = atoi(argv[2]);
     }	     
    if (argc > 3){ 
     kernel_select = atoi(argv[3]);
     }	
    
   // Select in which card the code is going to be executed
    hipSetDevice(d);
    
   // Select the image    
   char name[13];
    if (image_select == 1){
	    strcpy(name,"lena.txt");
      size_col = 512;
      size_row = 512;
	  }else if (image_select == 2){
	  	strcpy(name,"tran.txt");
      size_col = 1200;
      size_row = 800;
	  }else if(image_select == 3){
	    strcpy(name,"land.txt");
      size_col = 3840;
      size_row = 2160;
	  }
             
    // Allocate memory in the device
    int size = size_col*size_row*sizeof(double); 
    
    // Image
    hipHostAlloc(&a, size , hipHostMallocDefault);
    // Kernel Filter
    hipHostAlloc(&b, size_k , hipHostMallocDefault);
    // Result
    hipHostAlloc(&r, size , hipHostMallocDefault);       
        
    // Load the kernel filter
    int kernel_size = 3;
		double kernel[kernel_size][kernel_size];       
    srand(time(NULL));
    
    if(kernel_select == 1){
    	//CREATE A KERNEL FILTER WITH RANDOM VALUES
     double *temp3=b;
    	for(i = 0; i<kernel_size; i++){
	        for(j = 0; j<kernel_size; j++){
	            *temp3 = (rand()%100) /100.00;
               temp3++;
	        }
	    }      
  	}else if(kernel_select ==2){
  		//LAPLACIAN KERNEL FILTER            
      double kernel_vec[9] ={0,-1,0,-1,4,-1,0,-1,0};
      double *temp3 =b;
      for(i =0;i<9;i++){
        *temp3 = kernel_vec[i];
        temp3++;
      }         
  	}else if(kernel_select == 3){
  		//LOW PASS KERNEL FILTER  
  		 double kernel_vec[9] ={.1111,.1111,.1111,.1111,.1111,.1111,.1111,.1111,.1111};
       double *temp3 = b;
       for(i =0;i<9;i++){
          *temp3 = kernel_vec[i];
           temp3++;
       }  		
  	}        
  	         	
  	// Reading the filter from file
    FILE *img;
  	img = fopen(name,"r");
  	
  	if(img==NULL){
  		printf("Error loading image\n");
  		return 0;
  	}
   
    // Scan image  
  	for(i=0, tmp=a; i<size_col*size_row;++i){		
  		fscanf(img,"%lf",tmp++);  		
  	}
  	
    // Kernel execution    
    float sum = 0;
    float s_time = 0;
    
    for (i=0; i<noIterations;i++){ 
  	  t = convolutionDeviceKernel(a,b,r,size_col,size_row,kernel_size);
       sum += t;
    }
    
    s_time = sum/noIterations;
    // Print the execution time
  	printf("El tiempo de ejecucion es %f sg\n", s_time/1000.0); 	    
    
  	// Write the resulting image in a CSV file
  	FILE *nuevo;
  	nuevo = fopen("filtered_image.csv","w");
  	
  	if(nuevo==NULL){
  		printf("Error loading filter\n");
  		return 0;
  	}
			
  	for(i=1, tmp2=r; i<size_col*size_row;++i){  		
  		if(i%size_col*size_row==0 && i>0){
  			fprintf(nuevo,"%lf",*tmp2);
  			fprintf(nuevo,"\n");												
  		}else
  		{
  			fprintf(nuevo,"%lf,",*tmp2);      
  		}  								  			
  		tmp2++;		
  	}	   
  	fprintf(nuevo,"%lf",*tmp2);
   
    // Close the files img and nuevo
  	fclose(img);
  	fclose(nuevo);	
    // Free memory from the device
    hipHostFree(a);    	
 		hipHostFree(b);
		hipHostFree(r);
}
